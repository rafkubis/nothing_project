
#include <hip/hip_runtime.h>
#include <iostream>
constexpr size_t N{1000 * 7 + 6};

void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

__global__ void vector_add_cuda_old(float *out, float *a, float *b, int n) {
    for(size_t i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}


__global__ void vector_add_cuda(float *out, float *a, float *b, int n) {
    auto n_new = n / blockDim.x;
    auto mod2 = n % blockDim.x;
    auto i = threadIdx.x * n_new;
    auto j = i + n_new;
    if(threadIdx.x == blockDim.x - 1)
    {
        j+=mod2;
    }


    printf("new_new %d, j %d, i %d mod2 %d blockDin.x %d thread.x %d\n",
         n_new, j, i, mod2, blockDim.x, threadIdx.x);

    for(; i < j; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
    float *a_gpu, *b_gpu, *out_gpu; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    auto res = hipMalloc((void**)&a_gpu, sizeof(float) * N);
    std::cout << "Cuda malloc " << res << ", a_gpu:   "  << a_gpu << std::endl;
   res = hipMalloc((void**)&b_gpu, sizeof(float) * N);
    std::cout << "Cuda malloc " << res << ", b_gpu:   "  << b_gpu << std::endl;
   res = hipMalloc((void**)&out_gpu, sizeof(float) * N);
    std::cout << "Cuda malloc " << res << ", out_gpu: "  << out_gpu << std::endl;

    // Initialize array
    for(int i = 0; i < N; i++)
    {
        a[i] = 1.0f; b[i] = 2.0f;
    }
    res =   hipMemcpy(a_gpu, a, sizeof(float) * N * 1,hipMemcpyKind::hipMemcpyHostToDevice);
    std::cout << "Cuda hipMemcpy " << res << ", a_gpu:   "  << a_gpu << std::endl;

    res = hipMemcpy(b_gpu, b, sizeof(float) * N,hipMemcpyKind::hipMemcpyHostToDevice);
    std::cout << "Cuda hipMemcpy " << res << ", b_gpu:   "  << b_gpu << std::endl;


    // Main function
    //vector_add(out, a, b, N);

    vector_add_cuda<<<2,25>>>(out_gpu, a_gpu, b_gpu, N);

    
   res = hipMemcpy(out, out_gpu, sizeof(float) * N * 1, hipMemcpyKind::hipMemcpyDeviceToHost);
   std::cout << "Cuda hipMemcpy " << res << ", out_gpu: "  << out_gpu << std::endl;



    std::cout << out[255] << std::endl;
    std::cout << out[256] << std::endl;
    std::cout << out[257] << std::endl;
    std::cout << out[N-257] << std::endl;
    std::cout << out[N-258] << std::endl;
    std::cout << out[N-4] << std::endl;
    std::cout << out[N-3] << std::endl;
    std::cout << out[N-2] << std::endl;
    std::cout << out[N-1] << std::endl;
}